
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <iostream>
class Timer
{
private:
    /* data */
    std::chrono::time_point<std::chrono::steady_clock> start;

public:
    Timer(/* args */) : start(std::chrono::steady_clock::now()) {};
    ~Timer()
    {
        const std::chrono::duration<double> diff = std::chrono::steady_clock::now() - start;
        std::cout << "time:" << diff.count() << std::endl;
    };
};

__global__ void vector_add(float *out, float *a, float *b, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // int index = 0;
    // int stride = 1;

    for (int i = index; i < n; i += stride)
    {
        out[i] = a[i] + b[i];
    }
}
void helloFromGPU()
{
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    int N = 100000000;
    // Allocate memory
    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
        b[i] = i;
    }

    // Allocate device memory for a
    hipMalloc((void **)&d_out, sizeof(float) * N);
    hipMalloc((void **)&d_a, sizeof(float) * N);
    hipMalloc((void **)&d_b, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    // for (int i{}; i < 10; i++)
    //     std::cout << "Begin" << out[i] << std::endl;
    Timer *t = new Timer;

    // vector_add<<<1, 1>>>(d_out, d_a, d_b, N);
    vector_add<<<255, 255>>>(d_out, d_a, d_b, N);

    std::cout << hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    delete t;
    // for (int i{}; i < 10; i++)
    //     std::cout << "END" << out[i] << std::endl;

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
}
